#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "gpp_cuda_math.hpp"

#define OL_CUDA_STRINGIFY_EXPANSION_INNER(x) #x
#define OL_CUDA_STRINGIFY_EXPANSION(x) OL_CUDA_STRINGIFY_EXPANSION_INNER(x)
#define OL_CUDA_STRINGIFY_FILE_AND_LINE "(" __FILE__ ": " OL_CUDA_STRINGIFY_EXPANSION(__LINE__) ")"
#define OL_CUDA_ERROR_RETURN(X) do {if((X) != hipSuccess) {CudaError _err = {(X), OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__}; return _err;}} while (0);

namespace optimal_learning {

namespace {  // functions run on gpu device
/*
Special case of GeneralMatrixVectorMultiply.  As long as A has zeros in the strict upper-triangle,
GeneralMatrixVectorMultiply will work too (but take >= 2x as long).

Computes results IN-PLACE.
Avoids accessing the strict upper triangle of A.

Should be equivalent to BLAS call:
dtrmv('chol_var', trans, 'N', size_m, A, size_m, x, 1);

comment: This function is copied from gpp_linear_algebra.cpp
*/
__device__ void TriangularMatrixVectorMultiply_gpu(double const * __restrict__ A, int size_m, double * __restrict__ x) {
  double temp;
  A += size_m * (size_m-1);
  for (int j = size_m-1; j >= 0; --j) {  // i.e., j >= 0
    temp = x[j];
    for (int i = size_m-1; i >= j+1; --i) {
      // handles sub-diagonal contributions from j-th column
      x[i] += temp*A[i];
    }
    x[j] *= A[j];  // handles j-th on-diagonal component
    A -= size_m;
  }
}

/*
y = y - A * x (aka alpha = -1.0, beta = 1.0)

Computes matrix-vector product y = alpha * A * x + beta * y or y = alpha * A^T * x + beta * y
Since A is stored column-major, we need to treat the matrix-vector product as a weighted sum
of the columns of A, where x provides the weights.

That is, a matrix-vector product can be thought of as: (trans = 'T')
[  a_row1  ][   ]
[  a_row2  ][ x ]
[    ...   ][   ]
[  a_rowm  ][   ]
That is, y_i is the dot product of the i-th row of A with x.

OR the "dual" view: (trans = 'N')
[        |        |     |        ][ x_1 ]
[ a_col1 | a_col2 | ... | a_coln ][ ... ] = x_1*a_col1 + ... + x_n*a_coln
[        |        |     |        ][ x_n ]
That is, y is the weighted sum of columns of A.

Should be equivalent to BLAS call:
dgemv(trans='N', size_m, size_n, alpha, A, size_m, x, 1, beta, y, 1);

comment: This function is copied from gpp_linear_algebra.cpp
*/
__device__ void GeneralMatrixVectorMultiply_gpu(double const * __restrict__ A, double const * __restrict__ x, int size_m, int size_n, int lda, double * __restrict__ y) {
  double temp;
  for (int i = 0; i < size_n; ++i) {
    temp = -1.0 * x[i];
    for (int j = 0; j < size_m; ++j) {
      y[j] += A[j]*temp;
    }
    A += lda;
  }
}

// This inline function copies element from one array to the other, it also checks if index is out of bound before initiating the copy operation.
inline __device__ void copy_element(int index, int bound, double const * __restrict__ origin, double * __restrict__ destination) {
    if (index < bound) {
        destination[index] = origin[index];
    }
}

// EI_storage: A vector storing calculation result of EI from each thread
__global__ void EI_gpu(double const * __restrict__ chol_var, double const * __restrict__ mu, int num_union, int NUM_ITS, double best, unsigned int seed, double * __restrict__ EI_storage, double* __restrict__ gpu_random_number_EI, bool configure_for_test) {
  // copy mu, chol_var to shared memory mu_local & chol_var_local 
  // For multiple dynamically sized arrays in a single kernel, declare a single extern unsized array, and use
  // pointers into it to divide it into multiple arrays
  // refer to http://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
  extern __shared__ double storage[];
  double * chol_var_local = storage;
  double * mu_local = &chol_var_local[num_union * num_union];
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  const int loop_no = num_union * num_union / blockDim.x;
  for (int k = 0; k <= loop_no; ++k) {
    copy_element(k*blockDim.x+idx, num_union*num_union, chol_var, chol_var_local);
    copy_element(k*blockDim.x+idx, num_union, mu, mu_local);
  }
  __syncthreads();

  // MC start
  // RNG setup
  unsigned int local_seed = seed + IDX;
  hiprandState random_state;
  // seed a random number generator
  hiprand_init(local_seed, 0, 0, &random_state);

  double *normals = reinterpret_cast<double *>(malloc(sizeof(double)*num_union));
  double agg = 0.0;
  double improvement_this_step;
  double EI;

  for (int mc = 0; mc < NUM_ITS; ++mc) {
    improvement_this_step = 0.0;
    for (int i = 0; i < num_union; ++i) {
        normals[i] = hiprand_normal_double(&random_state);
        // If configure_for_test is ture, random numbers used in MC computations will be saved as output.
        // In fact we will let EI compuation on CPU use the same sequence of random numbers saved here,
        // so that EI compuation on CPU & GPU can be compared directly for unit test purpose.
        if (configure_for_test) {
            gpu_random_number_EI[IDX * NUM_ITS * num_union + mc * num_union + i] = normals[i];
        }
    }
    TriangularMatrixVectorMultiply_gpu(chol_var_local, num_union, normals);
    for (int i = 0; i < num_union; ++i) {
        EI = best - (mu_local[i] + normals[i]);
        improvement_this_step = fmax(EI, improvement_this_step);
    }
    agg += improvement_this_step;
  }
  EI_storage[IDX] = agg / static_cast<double>(NUM_ITS);
  free(normals);
}

// grad_EI_storage[dim][num_to_sample][num_threads]: A vector storing result of grad_EI from each thread
__global__ void grad_EI_gpu(double const * __restrict__ mu, double const * __restrict__ chol_var, double const * __restrict__ grad_mu, double const * __restrict__ grad_chol_var, double best, int num_union, int num_to_sample, int dim, int NUM_ITS, unsigned int seed,  double * __restrict__ grad_EI_storage, double* __restrict__ gpu_random_number_gradEI, bool configure_for_test) {
  // copy mu, chol_var, grad_mu, grad_chol_var to shared memory
  extern __shared__ double storage[];
  double * mu_local = storage;
  double * chol_var_local = &mu_local[num_union];
  double * grad_mu_local = &chol_var_local[num_union * num_union];
  double * grad_chol_var_local = &grad_mu_local[num_to_sample * dim];
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  const int loop_no = num_to_sample * num_union * num_union * dim / blockDim.x;
  for (int k = 0; k <= loop_no; ++k) {
      copy_element(k*blockDim.x+idx, num_to_sample*num_union*num_union*dim, grad_chol_var, grad_chol_var_local);
      copy_element(k*blockDim.x+idx, num_union*num_union, chol_var, chol_var_local);
      copy_element(k*blockDim.x+idx, num_to_sample*dim, grad_mu, grad_mu_local);
      copy_element(k*blockDim.x+idx, num_union, mu, mu_local);
  }
  __syncthreads();

  int i, k, mc, winner;
  double EI, improvement_this_step;
  // RNG setup
  unsigned int local_seed = seed + IDX;
  hiprandState random_state;
  hiprand_init(local_seed, 0, 0, &random_state);
  double* normals = reinterpret_cast<double*>(malloc(sizeof(double) * num_union));
  double* normals_copy = reinterpret_cast<double*>(malloc(sizeof(double) * num_union));
  // initialize grad_EI_storage
  for (int i = 0; i < (num_to_sample * dim); ++i) {
      grad_EI_storage[IDX*num_to_sample*dim + i] = 0.0;
  }
  // MC step start
  for (mc = 0; mc < NUM_ITS; ++mc) {
      improvement_this_step = 0.0;
      winner = -1;
      for (i = 0; i < num_union; ++i) {
          normals[i] = hiprand_normal_double(&random_state);
          normals_copy[i] = normals[i];
            // If configure_for_test is ture, random numbers used in MC computations will be saved as output.
            // In fact we will let gradEI compuation on CPU use the same sequence of random numbers saved here,
            // so that gradEI compuation on CPU & GPU can be compared directly for unit test purpose.
          if (configure_for_test) {
              gpu_random_number_gradEI[IDX * NUM_ITS * num_union + mc * num_union + i] = normals[i];
          }
      }
      TriangularMatrixVectorMultiply_gpu(chol_var_local, num_union, normals);
      for (i = 0; i < num_union; ++i) {
          EI = best - (mu_local[i] + normals[i]);
          if (EI > improvement_this_step) {
              improvement_this_step = EI;
              winner = i;
          }
      }
      if (improvement_this_step > 0.0) {
          if (winner < num_to_sample) {
              for (k = 0; k < dim; ++k) {
                  grad_EI_storage[IDX*num_to_sample*dim + winner * dim + k] -= grad_mu_local[winner * dim + k];
              }
          }
          for (i = 0; i < num_to_sample; ++i) {   // derivative w.r.t ith point
              GeneralMatrixVectorMultiply_gpu(grad_chol_var_local + i*num_union*num_union*dim + winner*num_union*dim, normals_copy, dim, num_union, dim, grad_EI_storage + IDX*num_to_sample*dim + i*dim);
          }
      }
  }

  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI_storage[IDX*num_to_sample*dim + i] /= static_cast<double>(NUM_ITS);
  }
  free(normals);
  free(normals_copy);
}

}  // end unnamed namespace

CudaError cuda_allocate_mem_for_double_vector(int num_doubles, double** __restrict__ address_of_ptr_to_gpu_memory) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  int mem_size = num_doubles * sizeof(double);
  OL_CUDA_ERROR_RETURN(hipMalloc(reinterpret_cast<void**>(address_of_ptr_to_gpu_memory), mem_size))
  return _success;
}

void cuda_free_mem(double* __restrict__ ptr_to_gpu_memory) {
  hipFree(ptr_to_gpu_memory);
}

CudaError cuda_get_EI(double * __restrict__ mu, double * __restrict__ chol_var, double best, int num_union, double * __restrict__ gpu_mu, double * __restrict__ gpu_chol_var, double * __restrict__ gpu_EI_storage, unsigned int seed, int num_mc, double* __restrict__ ei_val, double* __restrict__ gpu_random_number_EI, double* __restrict__ random_number_EI, bool configure_for_test) {
  *ei_val = 0.0;
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  // We assign EI_block_no blocks and EI_thread_no threads/block for EI computation, so there are (EI_block_no * EI_thread_no) threads in total to execute kernel function in parallel
  dim3 threads(EI_thread_no);
  dim3 grid(EI_block_no);
  double EI_storage[EI_thread_no * EI_block_no];
  int NUM_ITS = num_mc / (EI_thread_no * EI_block_no) + 1;   // make sure NUM_ITS is always >= 1

  int mem_size_mu = num_union * sizeof(double);
  int mem_size_chol_var = num_union * num_union * sizeof(double);
  int mem_size_EI_storage = EI_thread_no * EI_block_no * sizeof(double);
  // copy mu, chol_var to GPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice))
  // execute kernel
  EI_gpu <<< grid, threads, num_union*sizeof(double)+num_union*num_union*sizeof(double) >>> (gpu_chol_var, gpu_mu, num_union, NUM_ITS, best, seed, gpu_EI_storage, gpu_random_number_EI, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError())
  // copy gpu_EI_storage back to CPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(EI_storage, gpu_EI_storage, mem_size_EI_storage, hipMemcpyDeviceToHost))
  // copy gpu_random_number_EI back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_EI = NUM_ITS * EI_thread_no * EI_block_no * num_union * sizeof(double);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_EI, gpu_random_number_EI, mem_size_random_number_EI, hipMemcpyDeviceToHost))
  }
  // average EI_storage
  double ave = 0.0;
  for (int i = 0; i < (EI_thread_no*EI_block_no); ++i) {
      ave += EI_storage[i];
  }
  *ei_val = ave / static_cast<double>(EI_thread_no*EI_block_no);
  return _success;
}

// grad_EI[dim][num_to_sample]
CudaError cuda_get_gradEI(double * __restrict__ mu, double * __restrict__ grad_mu, double * __restrict__ chol_var, double * __restrict__ grad_chol_var, double best, int num_union, int num_to_sample, int dim, double * __restrict__ gpu_mu, double * __restrict__ gpu_grad_mu, double * __restrict__ gpu_chol_var, double * __restrict__ gpu_grad_chol_var, double * __restrict__ gpu_grad_EI_storage, unsigned int seed, int num_mc, double * __restrict__ grad_EI, double* __restrict__ gpu_random_number_gradEI, double* __restrict__ random_number_gradEI, bool configure_for_test) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  double grad_EI_storage[num_to_sample * dim * gradEI_thread_no * gradEI_block_no];
  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI[i] = 0.0;
  }
  // We assign gradEI_block_no blocks and gradEI_thread_no threads/block for gradEI computation, so there are (gradEI_block_no * gradEI_thread_no) threads in total to execute kernel function in parallel
  dim3 threads(gradEI_thread_no);
  dim3 grid(gradEI_block_no);
  int NUM_ITS = num_mc / (gradEI_thread_no * gradEI_block_no) + 1;   // make sure NUM_ITS is always >= 1

  int mem_size_mu = num_union * sizeof(double);
  int mem_size_grad_mu = num_to_sample * dim * sizeof(double);
  int mem_size_chol_var = num_union * num_union *sizeof(double);
  int mem_size_grad_chol_var = num_to_sample * num_union * num_union * dim * sizeof(double);
  int mem_size_grad_EI_storage= gradEI_thread_no * gradEI_block_no * num_to_sample * dim * sizeof(double);

  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_mu, grad_mu, mem_size_grad_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_chol_var, grad_chol_var, mem_size_grad_chol_var, hipMemcpyHostToDevice))

  // execute kernel
  // inputs: gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, best, num_union, num_to_sample, dim, NUM_ITS, seed
  // output: gpu_grad_EI_storage
  grad_EI_gpu <<< grid, threads, mem_size_mu+mem_size_chol_var+mem_size_grad_mu+mem_size_grad_chol_var >>> (gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, best, num_union, num_to_sample, dim, NUM_ITS, seed, gpu_grad_EI_storage, gpu_random_number_gradEI, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError())

  OL_CUDA_ERROR_RETURN(hipMemcpy(grad_EI_storage, gpu_grad_EI_storage, mem_size_grad_EI_storage, hipMemcpyDeviceToHost))
  // copy gpu_random_number_gradEI back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_gradEI = NUM_ITS * gradEI_thread_no * gradEI_block_no * num_union * sizeof(double);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_gradEI, gpu_random_number_gradEI, mem_size_random_number_gradEI, hipMemcpyDeviceToHost))
  }

  // The code block below extracts grad_EI from grad_EI_storage, which is output from the function
  // "cuda_get_gradEI" run on gpu. The way to do that is for each component of grad_EI, we find all
  // the threads calculating the corresponding component and average over the threads.
  for (int n = 0; n < (gradEI_thread_no*gradEI_block_no); ++n) {
      for (int i = 0; i < num_to_sample*dim; ++i) {
          grad_EI[i] += grad_EI_storage[n*num_to_sample*dim + i];
      }
  }
  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI[i] /= static_cast<double>(gradEI_thread_no*gradEI_block_no);
  }
  return _success;
}

CudaError cuda_set_device(int devID) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  OL_CUDA_ERROR_RETURN(hipSetDevice(devID))
  return _success;
}

}    // end namespace optimal_learning

