#include "hip/hip_runtime.h"
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "gpp_cuda_math.hpp"

#define OL_CUDA_STRINGIFY_EXPANSION_INNER(x) #x
#define OL_CUDA_STRINGIFY_EXPANSION(x) OL_CUDA_STRINGIFY_EXPANSION_INNER(x)
#define OL_CUDA_STRINGIFY_FILE_AND_LINE "(" __FILE__ ": " OL_CUDA_STRINGIFY_EXPANSION(__LINE__) ")"
#define OL_CUDA_ERROR_RETURN(X) do {if((X) != hipSuccess) {CudaError _err = {(X), OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__}; return _err;}} while (0);

namespace optimal_learning {

namespace {  // functions run on gpu device
/*
Special case of GeneralMatrixVectorMultiply.  As long as A has zeros in the strict upper-triangle,
GeneralMatrixVectorMultiply will work too (but take >= 2x as long).

Computes results IN-PLACE.
Avoids accessing the strict upper triangle of A.

Should be equivalent to BLAS call:
dtrmv('chol_var', trans, 'N', size_m, A, size_m, x, 1);

comment: This function is copied from gpp_linear_algebra.cpp
*/
__device__ void CudaTriangularMatrixVectorMultiply(double const * __restrict__ A, int size_m, double * __restrict__ x) {
  double temp;
  A += size_m * (size_m-1);
  for (int j = size_m-1; j >= 0; --j) {  // i.e., j >= 0
    temp = x[j];
    for (int i = size_m-1; i >= j+1; --i) {
      // handles sub-diagonal contributions from j-th column
      x[i] += temp*A[i];
    }
    x[j] *= A[j];  // handles j-th on-diagonal component
    A -= size_m;
  }
}

/*
y = y - A * x (aka alpha = -1.0, beta = 1.0)

Computes matrix-vector product y = alpha * A * x + beta * y or y = alpha * A^T * x + beta * y
Since A is stored column-major, we need to treat the matrix-vector product as a weighted sum
of the columns of A, where x provides the weights.

That is, a matrix-vector product can be thought of as: (trans = 'T')
[  a_row1  ][   ]
[  a_row2  ][ x ]
[    ...   ][   ]
[  a_rowm  ][   ]
That is, y_i is the dot product of the i-th row of A with x.

OR the "dual" view: (trans = 'N')
[        |        |     |        ][ x_1 ]
[ a_col1 | a_col2 | ... | a_coln ][ ... ] = x_1*a_col1 + ... + x_n*a_coln
[        |        |     |        ][ x_n ]
That is, y is the weighted sum of columns of A.

Should be equivalent to BLAS call:
dgemv(trans='N', size_m, size_n, alpha, A, size_m, x, 1, beta, y, 1);

comment: This function is copied from gpp_linear_algebra.cpp
*/
__device__ void CudaGeneralMatrixVectorMultiply(double const * __restrict__ A, double const * __restrict__ x, int size_m, int size_n, int lda, double * __restrict__ y) {
  double temp;
  for (int i = 0; i < size_n; ++i) {
    temp = -1.0 * x[i];
    for (int j = 0; j < size_m; ++j) {
      y[j] += A[j]*temp;
    }
    A += lda;
  }
}

// This inline function copies element from one array to the other, it also checks if index is out of bound before initiating the copy operation.
inline __device__ void CudaCopyElement(int index, int bound, double const * __restrict__ origin, double * __restrict__ destination) {
    if (index < bound) {
        destination[index] = origin[index];
    }
}

// EI_storage: A vector storing calculation result of EI from each thread
__global__ void CudaComputeEIGpu(double const * __restrict__ chol_var, double const * __restrict__ mu, int num_union, int num_iteration, double best, unsigned int seed, double * __restrict__ EI_storage, double* __restrict__ gpu_random_number_EI, bool configure_for_test) {
  // copy mu, chol_var to shared memory mu_local & chol_var_local 
  // For multiple dynamically sized arrays in a single kernel, declare a single extern unsized array, and use
  // pointers into it to divide it into multiple arrays
  // refer to http://devblogs.nvidia.com/parallelforall/using-shared-memory-cuda-cc/
  extern __shared__ double storage[];
  double * chol_var_local = storage;
  double * mu_local = &chol_var_local[num_union * num_union];
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  const int loop_no = num_union * num_union / blockDim.x;
  for (int k = 0; k <= loop_no; ++k) {
    CudaCopyElement(k*blockDim.x+idx, num_union*num_union, chol_var, chol_var_local);
    CudaCopyElement(k*blockDim.x+idx, num_union, mu, mu_local);
  }
  __syncthreads();
  double * ei_random_normal_container = &mu_local[num_union];
  // double *normals = reinterpret_cast<double *>(malloc(sizeof(double)*num_union));
  // normals[0] = 1;

  // MC start
  // RNG setup
  unsigned int local_seed = seed + IDX;
  hiprandState random_state;
  // seed a random number generator
  hiprand_init(local_seed, 0, 0, &random_state);

  double agg = 0.0;
  double improvement_this_step;
  double EI;

  for (int mc = 0; mc < num_iteration; ++mc) {
    improvement_this_step = 0.0;
    for (int i = 0; i < num_union; ++i) {
        ei_random_normal_container[idx * num_union + i] = hiprand_normal_double(&random_state);
        // If configure_for_test is ture, random numbers used in MC computations will be saved as output.
        // In fact we will let EI compuation on CPU use the same sequence of random numbers saved here,
        // so that EI compuation on CPU & GPU can be compared directly for unit test purpose.
        if (configure_for_test) {
            gpu_random_number_EI[IDX * num_iteration * num_union + mc * num_union + i] = ei_random_normal_container[idx * num_union + i];
        }
    }

    CudaTriangularMatrixVectorMultiply(chol_var_local, num_union, ei_random_normal_container + idx * num_union);

    for (int i = 0; i < num_union; ++i) {
        EI = best - (mu_local[i] + ei_random_normal_container[idx * num_union + i]);
        improvement_this_step = fmax(EI, improvement_this_step);
    }
    agg += improvement_this_step;
  }
  EI_storage[IDX] = agg / static_cast<double>(num_iteration);
  // free(normals);
}

// grad_EI_storage[dim][num_to_sample][num_threads]: A vector storing result of grad_EI from each thread
__global__ void CudaComputeGradEIGpu(double const * __restrict__ mu, double const * __restrict__ chol_var, double const * __restrict__ grad_mu, double const * __restrict__ grad_chol_var, double best, int num_union, int num_to_sample, int dim, int num_iteration, unsigned int seed,  double * __restrict__ grad_EI_storage, double* __restrict__ gpu_random_number_gradEI, bool configure_for_test) {
  // copy mu, chol_var, grad_mu, grad_chol_var to shared memory
  extern __shared__ double storage[];
  double * mu_local = storage;
  double * chol_var_local = &mu_local[num_union];
  double * grad_mu_local = &chol_var_local[num_union * num_union];
  double * grad_chol_var_local = &grad_mu_local[num_to_sample * dim];
  const int idx = threadIdx.x;
  const int IDX = threadIdx.x + blockDim.x * blockIdx.x;
  const int loop_no = num_to_sample * num_union * num_union * dim / blockDim.x;
  for (int k = 0; k <= loop_no; ++k) {
      CudaCopyElement(k*blockDim.x+idx, num_to_sample*num_union*num_union*dim, grad_chol_var, grad_chol_var_local);
      CudaCopyElement(k*blockDim.x+idx, num_union*num_union, chol_var, chol_var_local);
      CudaCopyElement(k*blockDim.x+idx, num_to_sample*dim, grad_mu, grad_mu_local);
      CudaCopyElement(k*blockDim.x+idx, num_union, mu, mu_local);
  }
  __syncthreads();
  double * grad_ei_random_normal_container = &grad_chol_var_local[num_union * num_union * num_to_sample * dim];
  // double* normals = reinterpret_cast<double*>(malloc(sizeof(double) * num_union));
  // double* normals_copy = reinterpret_cast<double*>(malloc(sizeof(double) * num_union));
  // normals[0] = 1;
  // normals_copy[0] = 1;

  int i, k, mc, winner;
  double EI, improvement_this_step;
  // RNG setup
  unsigned int local_seed = seed + IDX;
  hiprandState random_state;
  hiprand_init(local_seed, 0, 0, &random_state);
  // initialize grad_EI_storage
  for (int i = 0; i < (num_to_sample * dim); ++i) {
      grad_EI_storage[IDX*num_to_sample*dim + i] = 0.0;
  }
  // MC step start
  for (mc = 0; mc < num_iteration; ++mc) {
      improvement_this_step = 0.0;
      winner = -1;
      for (i = 0; i < num_union; ++i) {
          grad_ei_random_normal_container[idx * num_union * 2 + i] = hiprand_normal_double(&random_state);
          grad_ei_random_normal_container[idx * num_union * 2 + num_union + i] = grad_ei_random_normal_container[idx * num_union * 2 + i];
            // If configure_for_test is ture, random numbers used in MC computations will be saved as output.
            // In fact we will let gradEI compuation on CPU use the same sequence of random numbers saved here,
            // so that gradEI compuation on CPU & GPU can be compared directly for unit test purpose.
          if (configure_for_test) {
              gpu_random_number_gradEI[IDX * num_iteration * num_union + mc * num_union + i] = grad_ei_random_normal_container[idx * num_union * 2 + i];
          }
      }
      CudaTriangularMatrixVectorMultiply(chol_var_local, num_union, grad_ei_random_normal_container + idx * num_union * 2);
      for (i = 0; i < num_union; ++i) {
          EI = best - (mu_local[i] + grad_ei_random_normal_container[idx * num_union * 2 + i]);
          if (EI > improvement_this_step) {
              improvement_this_step = EI;
              winner = i;
          }
      }
      if (improvement_this_step > 0.0) {
          if (winner < num_to_sample) {
              for (k = 0; k < dim; ++k) {
                  grad_EI_storage[IDX*num_to_sample*dim + winner * dim + k] -= grad_mu_local[winner * dim + k];
              }
          }
          for (i = 0; i < num_to_sample; ++i) {   // derivative w.r.t ith point
              CudaGeneralMatrixVectorMultiply(grad_chol_var_local + i*num_union*num_union*dim + winner*num_union*dim, grad_ei_random_normal_container + idx * num_union * 2 + num_union, dim, num_union, dim, grad_EI_storage + IDX*num_to_sample*dim + i*dim);
          }
      }
  }

  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI_storage[IDX*num_to_sample*dim + i] /= static_cast<double>(num_iteration);
  }
  // free(normals);
  // free(normals_copy);
}
}  // end unnamed namespace

CudaError CudaAllocateMemForDoubleVector(int num_doubles, double** __restrict__ address_of_ptr_to_gpu_memory) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  int mem_size = num_doubles * sizeof(double);
  OL_CUDA_ERROR_RETURN(hipMalloc(reinterpret_cast<void**>(address_of_ptr_to_gpu_memory), mem_size))
  return _success;
}

void CudaFreeMem(double* __restrict__ ptr_to_gpu_memory) {
  hipFree(ptr_to_gpu_memory);
}

CudaError CudaGetEI(double * __restrict__ mu, double * __restrict__ chol_var, double best, int num_union, double * __restrict__ gpu_mu, double * __restrict__ gpu_chol_var, double * __restrict__ gpu_EI_storage, unsigned int seed, int num_mc, double* __restrict__ ei_val, double* __restrict__ gpu_random_number_EI, double* __restrict__ random_number_EI, bool configure_for_test) {
  *ei_val = 0.0;
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  // We assign ei_block_no blocks and ei_thread_no threads/block for EI computation, so there are (ei_block_no * ei_thread_no) threads in total to execute kernel function in parallel
  dim3 threads(ei_thread_no);
  dim3 grid(ei_block_no);
  double EI_storage[ei_thread_no * ei_block_no];
  int num_iteration = num_mc / (ei_thread_no * ei_block_no) + 1;   // make sure num_iteration is always >= 1

  int mem_size_mu = num_union * sizeof(double);
  int mem_size_chol_var = num_union * num_union * sizeof(double);
  int mem_size_EI_storage = ei_thread_no * ei_block_no * sizeof(double);
  // copy mu, chol_var to GPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice))
  // execute kernel
  CudaComputeEIGpu <<< grid, threads, num_union*sizeof(double)+num_union*num_union*sizeof(double)+num_union*ei_thread_no*sizeof(double) >>> (gpu_chol_var, gpu_mu, num_union, num_iteration, best, seed, gpu_EI_storage, gpu_random_number_EI, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError())
  // copy gpu_EI_storage back to CPU
  OL_CUDA_ERROR_RETURN(hipMemcpy(EI_storage, gpu_EI_storage, mem_size_EI_storage, hipMemcpyDeviceToHost))
  // copy gpu_random_number_EI back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_EI = num_iteration * ei_thread_no * ei_block_no * num_union * sizeof(double);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_EI, gpu_random_number_EI, mem_size_random_number_EI, hipMemcpyDeviceToHost))
  }
  // average EI_storage
  double ave = 0.0;
  for (int i = 0; i < (ei_thread_no*ei_block_no); ++i) {
      ave += EI_storage[i];
  }
  *ei_val = ave / static_cast<double>(ei_thread_no*ei_block_no);
  return _success;
}

// grad_EI[dim][num_to_sample]
CudaError CudaGetGradEI(double * __restrict__ mu, double * __restrict__ grad_mu, double * __restrict__ chol_var, double * __restrict__ grad_chol_var, double best, int num_union, int num_to_sample, int dim, double * __restrict__ gpu_mu, double * __restrict__ gpu_grad_mu, double * __restrict__ gpu_chol_var, double * __restrict__ gpu_grad_chol_var, double * __restrict__ gpu_grad_EI_storage, unsigned int seed, int num_mc, double * __restrict__ grad_EI, double* __restrict__ gpu_random_number_gradEI, double* __restrict__ random_number_gradEI, bool configure_for_test) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};

  double grad_EI_storage[num_to_sample * dim * grad_ei_thread_no * grad_ei_block_no];
  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI[i] = 0.0;
  }
  // We assign grad_ei_block_no blocks and grad_ei_thread_no threads/block for gradEI computation, so there are (grad_ei_block_no * grad_ei_thread_no) threads in total to execute kernel function in parallel
  dim3 threads(grad_ei_thread_no);
  dim3 grid(grad_ei_block_no);
  int num_iteration = num_mc / (grad_ei_thread_no * grad_ei_block_no) + 1;   // make sure num_iteration is always >= 1

  int mem_size_mu = num_union * sizeof(double);
  int mem_size_grad_mu = num_to_sample * dim * sizeof(double);
  int mem_size_chol_var = num_union * num_union *sizeof(double);
  int mem_size_grad_chol_var = num_to_sample * num_union * num_union * dim * sizeof(double);
  int mem_size_grad_EI_storage= grad_ei_thread_no * grad_ei_block_no * num_to_sample * dim * sizeof(double);

  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_mu, mu, mem_size_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_mu, grad_mu, mem_size_grad_mu, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_chol_var, chol_var, mem_size_chol_var, hipMemcpyHostToDevice))
  OL_CUDA_ERROR_RETURN(hipMemcpy(gpu_grad_chol_var, grad_chol_var, mem_size_grad_chol_var, hipMemcpyHostToDevice))

  // execute kernel
  // inputs: gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, best, num_union, num_to_sample, dim, num_iteration, seed
  // output: gpu_grad_EI_storage
  CudaComputeGradEIGpu <<< grid, threads, mem_size_mu+mem_size_chol_var+mem_size_grad_mu+mem_size_grad_chol_var+num_union*grad_ei_thread_no*2*sizeof(double) >>> (gpu_mu, gpu_chol_var, gpu_grad_mu, gpu_grad_chol_var, best, num_union, num_to_sample, dim, num_iteration, seed, gpu_grad_EI_storage, gpu_random_number_gradEI, configure_for_test);
  OL_CUDA_ERROR_RETURN(hipPeekAtLastError())

  OL_CUDA_ERROR_RETURN(hipMemcpy(grad_EI_storage, gpu_grad_EI_storage, mem_size_grad_EI_storage, hipMemcpyDeviceToHost))
  // copy gpu_random_number_gradEI back to CPU if configure_for_test is on
  if (configure_for_test) {
      int mem_size_random_number_gradEI = num_iteration * grad_ei_thread_no * grad_ei_block_no * num_union * sizeof(double);
      OL_CUDA_ERROR_RETURN(hipMemcpy(random_number_gradEI, gpu_random_number_gradEI, mem_size_random_number_gradEI, hipMemcpyDeviceToHost))
  }

  // The code block below extracts grad_EI from grad_EI_storage, which is output from the function
  // "CudaGetGradEI" run on gpu. The way to do that is for each component of grad_EI, we find all
  // the threads calculating the corresponding component and average over the threads.
  for (int n = 0; n < (grad_ei_thread_no*grad_ei_block_no); ++n) {
      for (int i = 0; i < num_to_sample*dim; ++i) {
          grad_EI[i] += grad_EI_storage[n*num_to_sample*dim + i];
      }
  }
  for (int i = 0; i < num_to_sample*dim; ++i) {
      grad_EI[i] /= static_cast<double>(grad_ei_thread_no*grad_ei_block_no);
  }
  return _success;
}

CudaError CudaSetDevice(int devID) {
  CudaError _success = {hipSuccess, OL_CUDA_STRINGIFY_FILE_AND_LINE, __func__};
  OL_CUDA_ERROR_RETURN(hipSetDevice(devID))
  OL_CUDA_ERROR_RETURN(hipDeviceSetCacheConfig(hipFuncCachePreferShared))
  return _success;
}

}    // end namespace optimal_learning

